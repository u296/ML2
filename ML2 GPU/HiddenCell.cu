#include "HiddenCell.cuh"

namespace ML2
{
	namespace Cells
	{
		ML2::Cells::HiddenCell::HiddenCell(std::vector<ML2::Bases::Cell *> inputCells, std::vector<double> weights, double(*activationFunction)(double))
		{
			m_inputCells			= inputCells;
			m_weights				= weights;
			m_activationFunction	= activationFunction;
		}
	}
}
